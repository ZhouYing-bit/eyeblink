#include "hip/hip_runtime.h"
#include "ica/fastica/kernels.h"

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void __global__ fica_sumAbs( float *d_sum, float *d_X )
{
  // Based on the "Optimizing Parallel Reduction in CUDA" paper by Mark Harris.
  extern __shared__ float s_data[];

  unsigned int row = threadIdx.y;
  unsigned int col = 0;

  float sum = 0.0f;

  #pragma unroll
  for (col = 0; col < blockDim.y; col++) {
    sum += fabsf( d_X[ col * blockDim.y + row ] );
  }

  s_data[row] = sum;
  __syncthreads();

  if (blockDim.y >= 256) { if (row < 128) { s_data[row] += s_data[row + 128]; }}
  __syncthreads();

  if (blockDim.y >= 128) { if (row <  64) { s_data[row] += s_data[row +  64]; }}
  __syncthreads();

  if (blockDim.y >=  64) { if (row <  32) { s_data[row] += s_data[row +  32]; }}
  if (blockDim.y >=  32) { if (row <  16) { s_data[row] += s_data[row +  16]; }}
  if (blockDim.y >=  16) { if (row <   8) { s_data[row] += s_data[row +   8]; }}

  if (row < 8) {
    s_data[row] += s_data[row + 4];
    s_data[row] += s_data[row + 2];
    s_data[row] += s_data[row + 1];
  }

  if (row == 0) {
    *d_sum = s_data[0];
  }
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void __global__ fica_scaleMatrix( float *d_X, float *alpha )
{
  unsigned int idx = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y +
                     threadIdx.y + threadIdx.x;
  d_X[idx] = d_X[idx] / alpha[0];
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void __global__ fica_tanh( float *d_ws, int ld )
{
  int idx = blockIdx.x * ld + threadIdx.y;
  d_ws[idx] = tanh( d_ws[idx] ); 
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void __global__ fica_tanhDer(float *d_ws, unsigned int ld, unsigned int n_cols)
{
  // Based on the "Optimizing Parallel Reduction in CUDA" paper by Mark Harris.
  float __shared__ s_data[256];

  const unsigned int tid = threadIdx.x;
  const unsigned int row = blockIdx.y;
  unsigned int i = tid;
  float accum;
  s_data[tid] = 0;

  // The big difference between this code and Mark Harris' code, is the change
  // from a simple summation, to the summing of (1 - x^2) for every element
  // 'x' in a row.
  while (i < n_cols) {
    accum = d_ws[ i * ld + row ];
    accum = 1.0f - accum * accum;
    s_data[tid] += accum;

    i += 256;
  }
  __syncthreads();

  if (tid < 128) {
    s_data[tid] += s_data[tid + 128];
  }
  __syncthreads();

  if (tid < 64) {
    s_data[tid] += s_data[tid + 64];
  }
  __syncthreads();

  if (tid < 32) {
    s_data[tid] += s_data[tid + 32];
    s_data[tid] += s_data[tid + 16];
    s_data[tid] += s_data[tid +  8];
    s_data[tid] += s_data[tid +  4];
    s_data[tid] += s_data[tid +  2];
    s_data[tid] += s_data[tid +  1];
  }

  if (tid == 0) {
    d_ws[ row ] = s_data[0];
  }
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void __global__ fica_cubeRule( float *d_wsum, float *d_ws,
                               unsigned int ld, unsigned int n_cols )
{
  float __shared__ s_data[256];

  const unsigned int tid = threadIdx.x;
  const unsigned int row = blockIdx.y;
  unsigned int i = tid;
  float val;

  s_data[tid] = 0.0f;

  while (i < n_cols) {
    val = d_ws[ i * ld + row ];
    s_data[tid] += 3.0f * val * val;
    d_ws[ i * ld + row ] = val * val * val;

    i += 256;
  }
  __syncthreads();

  if (tid < 128) {
    s_data[tid] += s_data[tid + 128];
  }
  __syncthreads();

  if (tid < 64) {
    s_data[tid] += s_data[tid + 64];
  }
  __syncthreads();

  if (tid < 32) {
    s_data[tid] += s_data[tid + 32];
    s_data[tid] += s_data[tid + 16];
    s_data[tid] += s_data[tid +  8];
    s_data[tid] += s_data[tid +  4];
    s_data[tid] += s_data[tid +  2];
    s_data[tid] += s_data[tid +  1];
  }

  if (tid == 0) {
    d_wsum[row] = s_data[0];
  }
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void __global__ fica_gaussRule( float *d_wsum, float *d_ws,
                                unsigned int ld, unsigned int n_cols )
{
  float __shared__ s_data[256];

  const unsigned int tid = threadIdx.x;
  const unsigned int row = blockIdx.y;
  unsigned int i = tid;
  unsigned int ws_idx;
  float wz, sqr, expo;

  s_data[tid] = 0.0f;

  while (i < n_cols) {
    ws_idx = i * ld + row;

    wz = d_ws[ ws_idx ];
    sqr = wz * wz;
    expo = exp( -sqr / 2.0f );

    s_data[tid] += (1.0f - sqr) * expo;
    d_ws[ ws_idx ] = wz * expo;

    i += 256;
  }
  __syncthreads();

  if (tid < 128) {
    s_data[tid] += s_data[tid + 128];
  }
  __syncthreads();

  if (tid < 64) {
    s_data[tid] += s_data[tid + 64];
  }
  __syncthreads();

  if (tid < 32) {
    s_data[tid] += s_data[tid + 32];
    s_data[tid] += s_data[tid + 16];
    s_data[tid] += s_data[tid +  8];
    s_data[tid] += s_data[tid +  4];
    s_data[tid] += s_data[tid +  2];
    s_data[tid] += s_data[tid +  1];
  }

  if (tid == 0) {
    d_wsum[row] = s_data[0];
  }
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void __global__ fica_wnext( float *d_w, float *d_wx, float *d_sums,
                            unsigned int ld, unsigned int n_cols )
{
  unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
  unsigned int idx = col * ld + row;

  d_w[idx] = (d_wx[idx] - d_sums[row] * d_w[idx]) / (float) n_cols;
}
