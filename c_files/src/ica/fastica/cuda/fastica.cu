#include "hip/hip_runtime.h"
#include "ica/ica.h"
#include "ica/aux.h"
#include "ica/setup.h"
#include "ica/fastica/contrast.h"
#include "ica/fastica/kernels.h"

#include <math.h>
#include <hipblas.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>

/**
 * Global variables setup in our initialization function. Setup of these
 * variables is an overhead that we shouldn't have to incur for every run of
 * the fastica() computation, since the values for these variables is dependent
 * on the ICA configuration parameters and nothing else.
 */
static Matrix _d_W[6] = {0}, _d_Z[2] = {0};     // Device matrices.
static Matrix _h_white = {0}, _h_dewhite = {0}; // Host matrices.
static GPUContFunc _contrast = NULL;            // Contrast function we apply.
static int _device = 0;                         // Which GPU device to use.

static NUMTYPE *_diag = NULL, *_eig_vals = NULL, *_mu_X = NULL; // Host memory.
static NUMTYPE *_d_sum = NULL;                  // Device memory.

static unsigned int _max_iter = 0;              // Max iterations to perform.

static dim3   _sum_grid(0);     // Grid size for summing kernel.
static dim3   _sum_block(0);    // Block size for summing kernel.
static size_t _sum_mem = 0;     // Size of shared memory for summing kernel.

static dim3 _scale_grid(0);     // Grid size for scaling kernel.
static dim3 _scale_block(0);    // Block size for scaling kernel.

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
int fastica_gpuInit( ICAParams *params )
{
  unsigned int i, size_diff, padded_w_size;

  _device = params->gpu_device;
  hipSetDevice( _device );
  cublasInit();

  _max_iter = params->max_iter;

  switch (params->contrast) {
    case NONLIN_CUBE:
      _contrast = gpu_negent_cube;
      break;
    case NONLIN_GAUSS:
      _contrast = gpu_negent_gauss;
      break;
    case NONLIN_TANH:
    default:
      _contrast = gpu_negent_tanh;
      break;
  }

  _sum_grid  = dim3( 1 );
  _sum_block = dim3( 1, params->num_var );
  _sum_mem   = sizeof(NUMTYPE) * params->num_var;

  //////////////////////////////////////////////////////////////////////////////
  // Allocate device memory.
  //////////////////////////////////////////////////////////////////////////////

  // We pad the ends of the 'W' matrices to ensure that we have allocated space
  // for a multiple of 256 floating point numbers. This makes thread allocation
  // later on easier on us.
  size_diff = 256 - ((params->num_var * params->num_var) % 256);
  if (size_diff == 256) { size_diff = 0; }

  padded_w_size = params->num_var * params->num_var + size_diff;
  _scale_grid  = dim3( padded_w_size / 256 );
  _scale_block = dim3( 256 );

  for (i = 0; i < sizeof(_d_W) / sizeof(Matrix); i++) {
    hipMalloc( (void**) &(_d_W[i].elem), sizeof(float) * padded_w_size );
    _d_W[i].rows = _d_W[i].ld  = params->num_var;
    _d_W[i].cols = _d_W[i].lag = params->num_var;
  }

  // We pad the end of the observation matrices for the same reason, though for
  // these guys we want each row to have a multiple of 256 elements. We don't
  // want those extra elements affecting things, though, so we set them to 0.
  size_diff = 256 - (params->num_obs % 256);
  if (size_diff == 256) {
    size_diff = 0;
  }

  for (i = 0; i < sizeof(_d_Z) / sizeof(Matrix); i++) {
    hipMalloc((void**) &(_d_Z[i].elem),
               sizeof(float) * params->num_var * (params->num_obs + size_diff));

    _d_Z[i].rows = _d_Z[i].ld = params->num_var;
    _d_Z[i].cols = params->num_obs;
    _d_Z[i].lag  = params->num_obs + size_diff;

    hipMemset( _d_Z[i].elem + params->num_var * params->num_obs, 0,
                size_diff * params->num_var * sizeof(float) );
  }

  hipMalloc( (void**) &_d_sum, sizeof(float) );

  //////////////////////////////////////////////////////////////////////////////
  // Allocate host memory.
  //////////////////////////////////////////////////////////////////////////////
  _h_dewhite.elem = (NUMTYPE*) malloc( sizeof(NUMTYPE) *
                                       params->num_var * params->num_var );
  _h_white.elem   = (NUMTYPE*) malloc( sizeof(NUMTYPE) *
                                       params->num_var * params->num_var );

  _h_dewhite.lag  = _h_white.lag  = params->num_var;
  _h_dewhite.cols = _h_white.cols = params->num_var;
  _h_dewhite.rows = _h_white.rows = params->num_var;
  _h_dewhite.ld   = _h_white.ld   = params->num_var;

  _eig_vals = (NUMTYPE*) malloc( sizeof(NUMTYPE) * params->num_var );
  _diag     = (NUMTYPE*) malloc( sizeof(NUMTYPE) * params->num_var );
  _mu_X     = (NUMTYPE*) malloc( sizeof(NUMTYPE) * params->num_var );

  // Return that everything went OK.
  // TODO: check for CUDA errors.
  return 1;
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void fastica_gpuShutdown()
{
  int i;

  //////////////////////////////////////////////////////////////////////////////
  // Free allocated memory and set everything to NULL/0.
  //////////////////////////////////////////////////////////////////////////////
  for (i = 0; i < sizeof(_d_W) / sizeof(Matrix); i++) {
    hipFree( _d_W[i].elem );
    _d_W[i].elem = NULL;
    _d_W[i].ld = _d_W[i].lag = _d_W[i].rows = _d_W[i].cols = 0;
  }

  for (i = 0; i < sizeof(_d_Z) / sizeof(Matrix); i++) {
    hipFree( _d_Z[i].elem );
    _d_Z[i].elem = NULL;
    _d_Z[i].ld = _d_Z[i].lag = _d_Z[i].rows = _d_Z[i].cols = 0;
  }

  hipFree( _d_sum );
  _d_sum = NULL;

  free( _h_dewhite.elem ); free( _h_white.elem ); free( _eig_vals );
  free( _diag ); free( _mu_X );

  _mu_X = _diag = _eig_vals = _h_dewhite.elem = _h_white.elem = NULL;
  _h_dewhite.ld = _h_dewhite.lag = _h_dewhite.rows = _h_dewhite.cols = 0;
  _h_white.ld = _h_white.lag = _h_white.rows = _h_white.cols = 0;

  _contrast = NULL;
  _device = _max_iter = 0;

  _sum_grid = _sum_block = dim3(0);
  _sum_mem = 0;

  _scale_grid = _scale_block = dim3(0);

  cublasShutdown();
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
unsigned int fastica_gpu( Matrix *W, Matrix *A, Matrix *S, float *mu_S,
                          Matrix const *X )
{
  int row, col, i, num_iter;
  float eig_inv_sqr, eig_sqr;

  //////////////////////////////////////////////////////////////////////////////
  // Make the observations zero-mean and find the matrix that will make those
  // zero-mean observations 'white' using the CPU. We only use the CPU here
  // because we don't have a handy function for computing eigenvalues (needed to
  // make the observations 'white') on the GPU.
  //////////////////////////////////////////////////////////////////////////////

  // Find the mean of each row of the observation matrix.
  remmean( _mu_X, S, X );

  //////////////////////////////////////////////////////////////////////////////
  // Find the whitening/dewhitening matrices.
  //////////////////////////////////////////////////////////////////////////////

  // We use the CPU for this because we don't have a convenient method for
  // getting the eigenvalue decomposition using the GPU.
  COVARIANCE( *W, *S );
  SYEV( *W, _eig_vals );

  for (col = 0; col < W->cols; col++) {
    eig_inv_sqr = 1.0 / sqrt( _eig_vals[col] );
    eig_sqr     = sqrt( _eig_vals[col] );

    for (row = 0; row < W->rows; row++) {
      i = col * W->rows + row;
      _h_white.elem[ row * W->rows + col ] = eig_inv_sqr * W->elem[i];
      _h_dewhite.elem[i] = eig_sqr * W->elem[i];
    }
  }

  //////////////////////////////////////////////////////////////////////////////
  // Copy the things we've calculated so far to the GPU. Almost all of the rest
  // of the ICA algorithm will now take place using the GPU.
  //////////////////////////////////////////////////////////////////////////////

  hipMemcpy( _d_W[0].elem, W->elem, sizeof(float) * W->rows * W->cols,
              hipMemcpyHostToDevice );
  hipMemcpy( _d_W[5].elem, _h_dewhite.elem, sizeof(float) * W->rows * W->cols,
              hipMemcpyHostToDevice );
  hipMemcpy( _d_W[4].elem, _h_white.elem,   sizeof(float) * W->rows * W->cols,
              hipMemcpyHostToDevice );
  hipMemcpy( _d_Z[1].elem, S->elem, sizeof(float) * X->rows * X->cols,
              hipMemcpyHostToDevice );

  // Whiten the zero-mean observations.
  CUBLAS_GEMM( _d_Z[0], _d_W[4], _d_Z[1] );

  //////////////////////////////////////////////////////////////////////////////
  // Now that the device holds the zero'd, whitened observations in _d_Z[0], and
  // the initial value for the unmixing matrix, W, in _d_W[0], we can begin
  // iterating.
  //////////////////////////////////////////////////////////////////////////////
  for (num_iter = 0; num_iter < _max_iter; num_iter++) {
    ////////////////////////////////////////////////////////////////////////////
    // Apply contrast to _d_W[x], store result in _d_W[0].
    ////////////////////////////////////////////////////////////////////////////
    _contrast( &(_d_W[0]), &(_d_Z[0]) );

    ////////////////////////////////////////////////////////////////////////////
    // Orthogonalize W.
    ////////////////////////////////////////////////////////////////////////////

    // Normalize W using the sum of the absolute value of its elements. This
    // ensures that all of its eigenvectors are less than one.
    fica_sumAbs<<< _sum_grid, _sum_block, _sum_mem >>>(
                   _d_sum, _d_W[0].elem );
    hipDeviceSynchronize();

    fica_scaleMatrix<<< _scale_grid, _scale_block >>>( _d_W[0].elem, _d_sum );
    hipDeviceSynchronize();

    do {
      // _d_W[3] = W * W'
      CUBLAS_GEMM_NT( _d_W[3], _d_W[0], _d_W[0] );

      // _d_W[0] = 1.5 * W - 0.5 * W * W' * W
      hipblasSgemm( 'N', 'N', W->rows, W->rows, W->rows,
                   -0.5,
                   _d_W[3].elem, W->rows,   // _d_W[3] == W * W'
                   _d_W[0].elem, W->rows,   // _d_W[0] == W
                   1.5,
                   _d_W[0].elem, W->rows ); 

      // _d_W[2] = _d_W[0] * _d_W[0]'
      CUBLAS_GEMM_NT( _d_W[2], _d_W[0], _d_W[0] );

      hipblasGetVector( W->rows, sizeof(float),
                       _d_W[2].elem, W->rows + 1,
                       _diag, 1 );
      for (i = 1; i < W->rows; i++) {
        _diag[0] += _diag[i];
      }
    } while ((float) W->rows - _diag[0] > 0.00001);
  }

  //////////////////////////////////////////////////////////////////////////////
  // We now have a best guess at an unmixing matrix. We need to finish up our
  // computations by computing the source signals, the source signal mixing
  // matrix, the unmixing matrix that will unmix the original, nonwhitened
  // observations, and the means of the source signals.
  //////////////////////////////////////////////////////////////////////////////

  // _d_W[5] holds the dewhitening matrix.
  // _d_W[4] holds the whitening matrix.
  // _d_W[0] holds the best guess at an unmixing matrix after our iterations.
  // _d_Z[0] holds the whitened, zero-mean observations.

  CUBLAS_GEMM_NT( _d_W[1], _d_W[5], _d_W[0] ); // _d_W[1] <- the mixing matrix
  CUBLAS_GEMM( _d_W[2], _d_W[0], _d_W[4] );    // _d_W[2] <- the unmixing matrix
  CUBLAS_GEMM( _d_Z[1], _d_W[0], _d_Z[0] );    // _d_Z[1] <- the source signals

  // Copy the results from the device back to the host.
  hipMemcpy( S->elem, _d_Z[1].elem, sizeof(float) * S->rows * S->cols,
              hipMemcpyDeviceToHost );
  hipMemcpy( W->elem, _d_W[0].elem, sizeof(float) * W->rows * W->cols,
              hipMemcpyDeviceToHost );
  hipMemcpy( A->elem, _d_W[1].elem, sizeof(float) * A->rows * A->cols,
              hipMemcpyDeviceToHost );

  GEMV( mu_S, *W, _mu_X );

  //////////////////////////////////////////////////////////////////////////////
  // And we're done!
  //////////////////////////////////////////////////////////////////////////////

  return num_iter;
}
