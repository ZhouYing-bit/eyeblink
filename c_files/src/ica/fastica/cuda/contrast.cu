#include "hip/hip_runtime.h"
#include "ica/fastica/contrast.h"

#include "ica/fastica/kernels.h"

#include <stdio.h>
#include <hipblas.h>

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void gpu_negent_tanh( Matrix *d_W, Matrix *d_Z )
{
  // Device workspace matrices.
  Matrix d_WS[2];
  d_WS[0].rows = d_Z->rows;
  d_WS[0].cols = d_Z->cols;
  d_WS[0].ld   = d_Z->rows;
  d_WS[0].lag  = d_Z->lag;

  d_WS[1].rows = d_W->rows;
  d_WS[1].cols = d_W->cols;
  d_WS[1].ld   = d_W->rows;
  d_WS[1].lag  = d_W->lag;

  // Allocate memory for the workspace matrices.
  hipMalloc( (void**) &(d_WS[0].elem),
              sizeof(float) * d_Z->ld * d_Z->lag );
  hipMalloc( (void**) &(d_WS[1].elem),
              sizeof(float) * d_W->ld * d_W->lag );

  // Make sure the d_WS[0] matrix has been zero-padded.
  hipMemset( d_WS[0].elem, 0x00, sizeof(float) * d_Z->ld * d_Z->lag );

  // d_WS[0] = W * Z
  CUBLAS_GEMM( d_WS[0], *d_W, *d_Z );

  // Calculate the tanh(.) of each element in the workspace, using one block
  // per column.
  // d_WS[0] = tanh( W * Z )
  dim3 dim_grid( d_WS[0].cols );
  dim3 dim_block( 1, d_WS[0].rows );
  fica_tanh<<< dim_grid, dim_block >>>( d_WS[0].elem, d_WS[0].rows );
  hipDeviceSynchronize();

  // d_WS[1] = tanh( W * Z ) * Z'
  CUBLAS_GEMM_NT( d_WS[1], d_WS[0], *d_Z );

  // Calculate (1 - tanh^2( W * Z )) and sum the rows of the resulting matrix.
  dim_grid   = dim3( 1, d_WS[0].rows );
  dim_block  = dim3( 256, 1 );
  fica_tanhDer<<< dim_grid, dim_block >>>( d_WS[0].elem, d_WS[0].ld,
                                           d_WS[0].cols );
  hipDeviceSynchronize();

  // Finish the application of the learning rule.
  dim_grid  = dim3( d_W->cols );
  dim_block = dim3( 1, d_W->rows );
  fica_wnext<<< dim_grid, dim_block >>>( d_W->elem, d_WS[1].elem, d_WS[0].elem,
                                         d_W->ld, d_Z->cols );
  hipDeviceSynchronize();

  hipFree( d_WS[0].elem );
  hipFree( d_WS[1].elem );
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void gpu_negent_cube( Matrix *d_W, Matrix *d_Z )
{
  Matrix d_WS;
  d_WS.rows = d_Z->rows;
  d_WS.cols = d_Z->cols;
  d_WS.ld   = d_Z->rows;
  d_WS.lag  = d_Z->lag;

  float *d_wsum;

  hipMalloc( (void**) &(d_WS.elem),
              sizeof(float) * d_Z->ld * d_Z->lag );
  hipMalloc( (void**) &d_wsum, sizeof(float) * d_Z->rows );

  hipMemset( d_WS.elem, 0, sizeof(float) * d_Z->ld * d_Z->lag );

  CUBLAS_GEMM( d_WS, *d_W, *d_Z );

  dim3 grid( 1, d_WS.rows );
  dim3 block( 256 );
  fica_cubeRule<<< grid, block >>>( d_wsum, d_WS.elem, d_WS.ld,
                                    d_WS.cols );
  hipDeviceSynchronize();

  CUBLAS_GEMM_NT( d_WS, d_WS, *d_Z );

  grid = dim3( 1, d_WS.rows );
  block = dim3( 256 );
  fica_wnext<<< grid, block >>>( d_W->elem, d_WS.elem, d_wsum, d_W->ld,
                                 d_Z->cols );
  hipDeviceSynchronize();

  hipFree( d_WS.elem );
  hipFree( d_wsum );
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void gpu_negent_gauss( Matrix *d_W, Matrix *d_Z )
{
  Matrix d_WS;
  d_WS.rows = d_Z->rows;
  d_WS.cols = d_Z->cols;
  d_WS.ld   = d_Z->rows;
  d_WS.lag  = d_Z->lag;

  float *d_wsum;

  hipMalloc( (void**) &(d_WS.elem),
              sizeof(float) * d_Z->ld * d_Z->lag );
  hipMalloc( (void**) &d_wsum, sizeof(float) * d_Z->rows );

  hipMemset( d_WS.elem, 0, sizeof(float) * d_Z->ld * d_Z->lag );

  CUBLAS_GEMM( d_WS, *d_W, *d_Z );

  dim3 grid( 1, d_WS.rows );
  dim3 block( 256 );
  fica_gaussRule<<< grid, block >>>( d_wsum, d_WS.elem, d_WS.ld,
                                     d_WS.cols );
  hipDeviceSynchronize();

  CUBLAS_GEMM_NT( d_WS, d_WS, *d_Z );

  grid = dim3( 1, d_WS.rows );
  block = dim3( 256 );
  fica_wnext<<< grid, block >>>( d_W->elem, d_WS.elem, d_wsum, d_W->ld,
                                 d_Z->cols );
  hipDeviceSynchronize();

  hipFree( d_WS.elem );
  hipFree( d_wsum );
}
