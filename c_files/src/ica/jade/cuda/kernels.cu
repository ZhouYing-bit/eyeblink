#include "hip/hip_runtime.h"
#include "ica/jade/kernels.h"

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void __global__ jade_identity( NUMTYPE *d_M, unsigned int ld )
{
  NUMTYPE val;
  if (threadIdx.x == blockIdx.x) {
    val = 1.0f;
  } else {
    val = 0.0f;
  }

  d_M[ blockIdx.x * ld + threadIdx.x ] = val;
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void __global__ jade_genCumulants( NUMTYPE *d_Q, NUMTYPE const *d_X,
                                   unsigned int q_ld, unsigned int x_ld,
                                   unsigned int num_var, unsigned int n_obs )
{
  // This kernel must be called with 256 threads per block.
  NUMTYPE __shared__ s_data[ 256 * 5 ];

  int index = blockIdx.y / num_var;

  const unsigned int var_k = blockIdx.x;
  const unsigned int var_l = blockIdx.y - index * num_var;

  const unsigned int s_prod = threadIdx.x + 4 * 256;
  const unsigned int q_idx  = blockIdx.y * q_ld + blockIdx.x;

  unsigned int var_i;
  unsigned int var_j;

  unsigned int col, t_idx;

  unsigned int s_iidx, s_jidx, s_kidx, s_lidx;
  unsigned int x_iidx, x_jidx, x_kidx, x_lidx;
  
  NUMTYPE l_prod, r_prod;

  unsigned int delta = 0;

  // Figure out the values for i and j for this matrix.
  while (index >= 0) {
    var_i = delta;
    var_j = index + delta;

    index -= (num_var - delta);

    delta++;
  }

  // Initialize the accumulators.
  s_data[s_prod] = 0.0f;

  // Read in the first groups of variable observations.
  for (col = 0; col < n_obs; col += 256) {
    // Read in values for variables i,j,k,l.
    s_iidx = threadIdx.x + 0 * 256; x_iidx = var_i * x_ld + threadIdx.x + col;
    s_jidx = threadIdx.x + 1 * 256; x_jidx = var_j * x_ld + threadIdx.x + col;
    s_kidx = threadIdx.x + 2 * 256; x_kidx = var_k * x_ld + threadIdx.x + col;
    s_lidx = threadIdx.x + 3 * 256; x_lidx = var_l * x_ld + threadIdx.x + col;

    s_data[s_iidx] = d_X[x_iidx];
    s_data[s_jidx] = d_X[x_jidx];
    s_data[s_kidx] = d_X[x_kidx];
    s_data[s_lidx] = d_X[x_lidx];

    // Calculate and accumulate the product of variables i,j,k,l. We compute the
    // product as ((i*j) * (k*l)) because i,j,k,l should be the same magnitude,
    // and this multiplication order should reduce rounding error.
    l_prod = s_data[s_iidx] * s_data[s_jidx];
    r_prod = s_data[s_kidx] * s_data[s_lidx];

    s_data[s_prod] += (l_prod * r_prod) / (NUMTYPE) n_obs;
  }
  __syncthreads();

  t_idx = threadIdx.x;

  // We now need to sum the accumulated products.
  if (t_idx < 128) { s_data[s_prod] += s_data[s_prod + 128]; }
  __syncthreads();

  if (t_idx <  64) { s_data[s_prod] += s_data[s_prod +  64]; }
  __syncthreads();

  if (t_idx <  32) {
    s_data[s_prod] += s_data[s_prod + 32];
    s_data[s_prod] += s_data[s_prod + 16];
    s_data[s_prod] += s_data[s_prod +  8];
    s_data[s_prod] += s_data[s_prod +  4];
    s_data[s_prod] += s_data[s_prod +  2];
    s_data[s_prod] += s_data[s_prod +  1];
  }

  // Record the final sum.
  if (t_idx == 0) {
    // Because of the whitening step, whenever the variable indices are equal to
    // each other, if all four are equal then the cumulant is equal to
    // E{Xi*Xi*Xi*Xi} - 3.0, otherwise, if the indices still form equal pairs,
    // the cumulant is equal to E{Xi*Xi*Xj*Xj} - 1.0. Otherwise, if at least two
    // of the indices differ, then the cumulant is simply equal to
    // E{Xi*Xj*Xk*Xl}. Before recording the calculated sum of products, we need
    // to apply any necessary DC offset.

    if (var_i == var_j && var_i == var_k && var_i == var_l) {
      d_Q[q_idx] = s_data[s_prod] - 3.0f; // i=j == k=l
    } else if (var_i == var_j && var_k == var_l) {
      d_Q[q_idx] = s_data[s_prod] - 1.0f; // i=j != k=l
    } else if (var_i == var_k && var_j == var_l) {
      d_Q[q_idx] = s_data[s_prod] - 1.0f; // i=k != j=l
    } else if (var_i == var_l && var_j == var_k) {
      d_Q[q_idx] = s_data[s_prod] - 1.0f; // i=l != j=k
    } else {
      d_Q[q_idx] = s_data[s_prod];
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void __global__ jade_rightRot( NUMTYPE *d_Q, NUMTYPE *d_rot,
                               unsigned int q_ld, unsigned int rot_ld,
                               unsigned int sequence, unsigned int num_var )
{
  // This kernel must be called with (num_var - (num_var + 1) / 2) blocks
  // per cumulant matrix, for a total of:
  //    (num_var - (num_var + 1) / 2 - 1) * (num_var * (num_var + 1)) / 2
  // blocks.
  // This kernel must be called with `num_var'x2 threads per block.
  NUMTYPE extern __shared__ q_cols[];
  NUMTYPE extern __shared__ r_cols[];
  NUMTYPE new_val, cosine, sine;

  const unsigned int blks_per_mat = num_var - (num_var + 1) / 2;
  const unsigned int q_mat        = blockIdx.x / blks_per_mat;
  const unsigned int q_base       = q_mat * num_var * q_ld;

  unsigned int p, q;
  jade_getPQ( &p, &q, sequence, blockIdx.x % blks_per_mat, num_var ); 

  const unsigned int s_p = 0 * num_var + threadIdx.x;
  const unsigned int s_q = 1 * num_var + threadIdx.x;

  // Fetch the columns we're operating on. The threadIdx.y value will be either
  // 0 or 1. If it's 0, the thread fetches a value from column p. If it's 1, the
  // thread fetches a value from column q.
  const unsigned int s_idx = threadIdx.y * num_var + threadIdx.x;
  const unsigned int q_idx = q_base +
                             (1 - threadIdx.y) * (p * q_ld) +
                             (    threadIdx.y) * (q * q_ld) +
                             threadIdx.x;

  q_cols[s_idx] = d_Q[q_idx];

  // Fetch the cosine and sine values from the rotation matrix.
  cosine = d_rot[p + p * rot_ld]; // row p, column, p
  sine   = d_rot[q + p * rot_ld]; // row q, column, p
  __syncthreads();

  new_val = (1-threadIdx.y) * (cosine * q_cols[s_p] +   sine * q_cols[s_q]) +
            (  threadIdx.y) * ( -sine * q_cols[s_p] + cosine * q_cols[s_q]);

  // Write the new values back.
  d_Q[q_idx] = new_val;
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void __device__ jade_getPQ( unsigned int *p, unsigned int *q,
                            unsigned int sequence, unsigned int pair,
                            unsigned int num_var )
{
  const unsigned int m = (num_var + 1) / 2;
  unsigned int temp;

  // The algorithm in the above paper provides for one-based indexing, but we
  // use zero-based, so this implementation will differ slightly.
  if (sequence < m) {
    *q = m - sequence + pair;

    if      (*q <= 2*m - 2*sequence - 1) { *p = 2*m - 2*sequence - *q - 1; }
    else if (*q <= 2*m -   sequence - 2) { *p = 4*m - 2*sequence - *q - 2; }
    else                                 { *p = num_var - 1; }
  } else {
    *q = 4*m - num_var - sequence + pair - 1;

    if      (*q <  2*m -   sequence)     { *p = num_var - 1; }
    else if (*q <= 4*m - 2*sequence - 2) { *p = 4*m - 2*sequence - *q - 2; }
    else                                 { *p = 6*m - 2*sequence - *q - 3; }
  }

  // Make sure that element (p,p) is 'higher' than element (q,q).
  if (*p > *q) { temp = *p; *p = *q; *q = temp; }
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void __global__ jade_anglesStepOne( NUMTYPE *d_vals, NUMTYPE const *d_Q,
                                    unsigned int q_ld,
                                    unsigned int seq, unsigned int num_var )
{
  // Called with one block per cumulant matrix and num_var/2 threads per block.
  unsigned int p, q;
  jade_getPQ( &p, &q, seq, threadIdx.x, num_var );

  const unsigned int q_mat  = blockIdx.x;
  const unsigned int q_base = q_mat * num_var * q_ld;

  const unsigned int q_ppidx = q_base + (p + p * q_ld); // row p, column p
  const unsigned int q_pqidx = q_base + (p + q * q_ld); // row p, column q
  const unsigned int q_qqidx = q_base + (q + q * q_ld); // row q, column q

  // Read in the elements (p,p), (p,q), and (q,q). Cumulant matrices are
  // symmetric, so element (p,q) equals element (q,p).
  const float q_pp = d_Q[q_ppidx];
  const float q_pq = d_Q[q_pqidx];
  const float q_qq = d_Q[q_qqidx];

  // Calculate the values from this cumulant matrix that contribute to the
  // rotation angle.
  const float diag_diff = q_pp - q_qq;
  const float off_diag  = 2.0f * q_pq;

  const float diag_sqr = diag_diff * diag_diff;
  const float off_sqr  = off_diag  * off_diag;
  const float cross    = off_diag  * diag_diff;

  // Record the computed values so that they can be summed by the next kernel.
  const unsigned int val_idx = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned int row_len = blockDim.x * gridDim.x;

  d_vals[ val_idx + row_len * 0 ] = diag_sqr;
  d_vals[ val_idx + row_len * 1 ] = off_sqr;
  d_vals[ val_idx + row_len * 2 ] = cross;
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void __global__ jade_anglesStepTwo( NUMTYPE *d_rot, NUMTYPE *d_vals,
                                    unsigned int rot_ld,
                                    unsigned int seq, unsigned int num_var )
{
  // Called with one block of 3 * (num_var / 2) threads and shared memory for
  // 3*(num_var/2) elements.

  NUMTYPE extern __shared__ sums[];

  const unsigned int num_cm   = (num_var * (num_var + 1)) / 2;
  const unsigned int val_base = (threadIdx.x/(num_var/2)) *
                                (num_var/2) * num_cm;
  const unsigned int pair     = threadIdx.x % (num_var / 2);

  // Start summing values.
  const unsigned int sum_idx = threadIdx.x;
  unsigned int i;

  sums[sum_idx] = 0.0f;
  NUMTYPE val;
  for (i = 0; i < num_cm; i++) {
    val = d_vals[ val_base + pair + i * (num_var/2) ];
    sums[sum_idx] += val;
  }
  __syncthreads();

  // Compute the final values used in the angle computations.
  NUMTYPE sqr_diff, cross_sum, theta, cosine, sine;
  unsigned int p,q;

  if (threadIdx.x < (num_var/2)) {
    sqr_diff  = sums[sum_idx] - sums[sum_idx + (num_var/2)];
    cross_sum = sums[sum_idx + (num_var/2) * 2] * 2.0f;

    theta = 0.5f * atan2( cross_sum, sqr_diff +
                          sqrt(sqr_diff * sqr_diff + cross_sum * cross_sum) );
    cosine = cos(theta);
    sine   = sin(theta);

    jade_getPQ( &p, &q, seq, pair, num_var );

    d_rot[ p + p * rot_ld ] = cosine; // row p, column p
    d_rot[ p + q * rot_ld ] = -sine;  // row p, column q
    d_rot[ q + p * rot_ld ] = sine;   // row q, column p
    d_rot[ q + q * rot_ld ] = cosine; // row q, column q
  }
}
