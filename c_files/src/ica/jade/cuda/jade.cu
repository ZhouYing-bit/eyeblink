#include "ica/ica.h"
#include "ica/aux.h"
#include "ica/setup.h"
#include "ica/jade/kernels.h"

#include <math.h>
#include <hipblas.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>

/**
 * Global variables setup in our initialization function. Setup of these
 * variables is an overhead that we shouldn't have to incur for every run of
 * the fastica() computation, since the values for these variables is dependent
 * on the ICA configuration parameters and nothing else.
 */

// The number of variables in the X matrix, the number of cumulant matrices
// that we will generate, the number of elements in a cumulant matrix, and the
// `m' parameter for the rotation matrix generation kernel.
static unsigned int _num_var = 0;
static unsigned int _num_cm = 0;
static unsigned int _num_elem = 0;
static unsigned int _m_param = 0;

// The size (in bytes) of a cumulant matrix.
static size_t _mat_size = 0;

// Device memory pointers.
static Matrix _d_Q = {0}, _d_Z[2] = {0}, _d_W[4] = {0};
static NUMTYPE *_d_vals = NULL;

// Host memory pointers.
static Matrix _h_white = {0}, _h_dewhite = {0}, _h_Z = {0};
static NUMTYPE *_h_Q = NULL, *_mu_X = NULL;

// Dimensions used by CUDA kernel functions.
static dim3 _grid_size(0), _block_size(0), _s1_grid(0), _s1_block(0);
static dim3 _rot_grid(0),  _rot_block(0),  _s2_grid(0), _s2_block(0);
static size_t _s2_mem = 0, _rot_mem = 0;

// Which GPU device to use.
static int _device = 0;

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
int jade_gpuInit( ICAParams *params )
{
  unsigned int i, size_diff, padded_X_size;

  _num_var  = params->num_var;
  _num_cm   = (_num_var * (_num_var + 1)) / 2;
  _num_elem = _num_var * _num_var;
  _m_param  = _num_var - (_num_var + 1) / 2;

  _mat_size = sizeof(NUMTYPE) * _num_elem;

  // Used to form cumulant matrices.
  _grid_size  = dim3( _num_var, _num_var * _num_cm, 1 );
  _block_size = dim3( 256 );

  // Used to create rotation matrix.
  _s1_grid = dim3( _num_cm, 1, 1 ); _s1_block = dim3( _num_var / 2, 1, 1 );

  // Used to create rotation matrix.
  _s2_grid = dim3( 1, 1, 1 ); _s2_block = dim3( 3 * (_num_var/2), 1, 1 );
  _s2_mem  = 3 * (_num_var/2) * sizeof(NUMTYPE);

  // Used to rotate cumulant matrices.
  _rot_grid = dim3( _m_param * _num_cm ); _rot_block = dim3( _num_var, 2 );
  _rot_mem  = 4 * _num_var * sizeof(NUMTYPE);

  // Initialize CUDA.
  _device = params->gpu_device;
  hipSetDevice( _device );

  //////////////////////////////////////////////////////////////////////////////
  // Allocate device memory.
  //////////////////////////////////////////////////////////////////////////////

  // Three values, per cumulant matrix, per pair of rows/columns.
  hipMalloc((void**) &_d_vals, 3 * _num_cm * (_num_var / 2) * sizeof(NUMTYPE));

  hipMalloc((void**) &(_d_Q.elem), _mat_size * _num_cm);
  _d_Q.ld  = _d_Q.rows = _num_var;
  _d_Q.lag = _d_Q.cols = _num_cm * _num_var;

  // We will pad the X matrix with zero rows so that the length of X is a
  // multiple of 256. This will give our CUDA kernels an easier time. We will
  // also be storing the transpose of X on the GPU, as this will allow for
  // memory coalescing later on.
  size_diff = 256 - (params->num_obs % 256);
  if (size_diff == 256) { size_diff = 0; }
  padded_X_size = _num_var * (params->num_obs + size_diff) * sizeof(NUMTYPE);

  for (i = 0; i < sizeof(_d_Z) / sizeof(Matrix); i++) {
    hipMalloc( (void**) &(_d_Z[i].elem), padded_X_size );

    _d_Z[i].cols = _d_Z[i].lag = _num_var;
    _d_Z[i].rows = params->num_obs;
    _d_Z[i].ld   = params->num_obs + size_diff;

    hipMemset( _d_Z[i].elem, 0, padded_X_size );
  }

  for (i = 0; i < sizeof(_d_W) / sizeof(Matrix); i++) {
    hipMalloc((void**)&(_d_W[i].elem), sizeof(NUMTYPE) * _num_var * _num_var );
    _d_W[i].rows = _d_W[i].ld  = _num_var;
    _d_W[i].cols = _d_W[i].lag = _num_var;
  }

  //////////////////////////////////////////////////////////////////////////////
  // Allocate host memory and initialize matrices.
  //////////////////////////////////////////////////////////////////////////////
  _h_dewhite.elem = (NUMTYPE*) malloc( sizeof(NUMTYPE) * _num_var * _num_var );
  _h_white.elem   = (NUMTYPE*) malloc( sizeof(NUMTYPE) * _num_var * _num_var );
  _h_Z.elem       = (NUMTYPE*) malloc( sizeof(NUMTYPE)*_d_Z[0].ld*_d_Z[0].lag);

  _h_dewhite.lag  = _h_white.lag  = _num_var;
  _h_dewhite.cols = _h_white.cols = _num_var;
  _h_dewhite.rows = _h_white.rows = _num_var;
  _h_dewhite.ld   = _h_white.ld   = _num_var;

  _h_Z.lag  = _d_Z[0].lag;
  _h_Z.cols = _d_Z[0].cols;
  _h_Z.rows = _d_Z[0].rows;
  _h_Z.ld   = _d_Z[0].ld;

  memset( _h_Z.elem, 0, _h_Z.ld * _h_Z.lag * sizeof(NUMTYPE) );

  _h_Q  = (NUMTYPE*) malloc( _mat_size * _num_cm );
  _mu_X = (NUMTYPE*) malloc( sizeof(NUMTYPE) * _num_var );

  // Return that everything went OK.
  // TODO: check for CUDA errors.
  return 1;
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void jade_gpuShutdown()
{
  int i;

  //////////////////////////////////////////////////////////////////////////////
  // Free allocated memory and set everything to NULL/0.
  //////////////////////////////////////////////////////////////////////////////
  for (i = 0; i < sizeof(_d_W) / sizeof(Matrix); i++) {
    hipFree( _d_W[i].elem );
    _d_W[i].elem = NULL;
    _d_W[i].ld = _d_W[i].lag = _d_W[i].rows = _d_W[i].cols = 0;
  }

  for (i = 0; i < sizeof(_d_Z) / sizeof(Matrix); i++) {
    hipFree( _d_Z[i].elem );
    _d_Z[i].elem = NULL;
    _d_Z[i].ld = _d_Z[i].lag = _d_Z[i].rows = _d_Z[i].cols = 0;
  }

  hipFree( _d_Q.elem );
  _d_Q.elem = NULL;
  _d_Q.ld = _d_Q.lag = _d_Q.rows = _d_Q.cols = 0;

  hipFree( _d_vals );
  _d_vals = NULL;

  free( _h_dewhite.elem ); free( _h_white.elem ); free( _h_Q ); free( _mu_X );
  _h_dewhite.elem = _h_white.elem = _h_Q = _mu_X = NULL;
  _h_dewhite.ld = _h_dewhite.lag = _h_dewhite.rows = _h_dewhite.cols = 0;
  _h_white.ld   = _h_white.lag   = _h_white.rows   = _h_white.cols = 0;

  _grid_size = _block_size = _s1_grid = _s1_block = dim3(0);
  _rot_grid  = _rot_block  = _s2_grid = _s2_block = dim3(0);
  _s2_mem = _rot_mem = 0;

  _device = 0;

  cublasShutdown();
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
unsigned int jade_gpu( Matrix *W, Matrix *A, Matrix *S, NUMTYPE *mu_S,
                       Matrix const *X )
{
  // The number of variables could probably be cut down, but we leave that to
  // the compiler. Readibility is the more important goal.
  unsigned int i, row, col, s_i, z_i, sweeps;

  //////////////////////////////////////////////////////////////////////////////
  // Make the observations zero-mean, remembering the observation means, so that
  // we can calculate the source signal means later.
  //////////////////////////////////////////////////////////////////////////////

  remmeanTranspose( _mu_X, &_h_Z, X );

  //////////////////////////////////////////////////////////////////////////////
  // Find the whitening/dewhitening matrices.
  //////////////////////////////////////////////////////////////////////////////

  // We use the CPU for this because we don't have a convenient method for
  // getting the eigenvalue decomposition using the GPU.
  computeWhiten( &_h_white, &_h_dewhite, &_h_Z, 1 );

  //////////////////////////////////////////////////////////////////////////////
  // Copy the things we've calculated so far to the GPU. Almost all of the rest
  // of the ICA algorithm will now take place using the GPU.
  //////////////////////////////////////////////////////////////////////////////

  hipMemcpy( _d_W[0].elem, _h_dewhite.elem, sizeof(NUMTYPE) * W->rows * W->cols,
              hipMemcpyHostToDevice );
  hipMemcpy( _d_W[1].elem, _h_white.elem,   sizeof(NUMTYPE) * W->rows * W->cols,
              hipMemcpyHostToDevice );
  hipMemcpy( _d_Z[0].elem, _h_Z.elem, sizeof(NUMTYPE) * _h_Z.ld * _h_Z.lag,
              hipMemcpyHostToDevice );

  // Whiten the zero-mean observations, remembering that the _d_Z matrices are
  // stored as their transpositions, i.e. we want Y = W * X, but both Y and X
  // must be stored as their transposes, so we end up calculating Y' = X' * W'
  CUBLAS_GEMM_NT( _d_Z[1], _d_Z[0], _d_W[1] );

  //////////////////////////////////////////////////////////////////////////////
  // Form the cumulant matrices.
  //////////////////////////////////////////////////////////////////////////////
  jade_genCumulants<<< _grid_size, _block_size >>>( _d_Q.elem,
                                                  _d_Z[1].elem,
                                                  _num_var,
                                                  _d_Z[1].ld,
                                                  _d_Z[1].cols,
                                                  _d_Z[1].rows );

  // Initialize the accumulated rotation matrix to identity.
  jade_identity<<< _num_var, _num_var >>>( _d_W[3].elem, _d_W[3].ld );

  hipDeviceSynchronize();

  //////////////////////////////////////////////////////////////////////////////
  // Begin performing Jacobi sweeps in an attempt to diagonalize all cumulant
  // matrices simulaneously.
  //////////////////////////////////////////////////////////////////////////////
  for (sweeps = 0; sweeps < 100; sweeps++) {
    for (i = 1; i < 2 * ((_num_var + 1) / 2); i++) {
      // Reset the matrix we use to store rotation values during each sweep.
      jade_identity<<< _num_var, _num_var >>>( _d_W[2].elem, _d_W[2].ld );
      hipDeviceSynchronize();

      // Calculate the rotation matrix to apply.
      jade_anglesStepOne<<< _s1_grid, _s1_block >>>( _d_vals, _d_Q.elem, _d_Q.ld,
                                                   i, _num_var );
      hipDeviceSynchronize();

      jade_anglesStepTwo<<< _s2_grid, _s2_block, _s2_mem >>>(
                            _d_W[2].elem, _d_vals, _d_W[2].ld, i, _num_var );
      hipDeviceSynchronize();

      // First, left multiple the rotation matrix to every cumulant matrix. We
      // can use the CUBLAS function for this.
      CUBLAS_GEMM_TN( _d_Q, _d_W[2], _d_Q );
      hipDeviceSynchronize();

      // Next, right multiple the rotation matrix to every cumulant matrix.
      jade_rightRot<<< _rot_grid, _rot_block, _rot_mem >>>( _d_Q.elem,
                                                         _d_W[2].elem,
                                                         _d_Q.ld,
                                                         _d_W[2].ld,
                                                         i, _num_var );
      // While that's going on, accumulate the rotation matrix.
      jade_rightRot<<< _m_param, _rot_block, _rot_mem >>>(
                                                      _d_W[3].elem, _d_W[2].elem,
                                                      _d_W[3].ld,   _d_W[2].ld,
                                                      i, _num_var );
      hipDeviceSynchronize();
    }
  }

  //////////////////////////////////////////////////////////////////////////////
  // We now have a best guess at an unmixing matrix. We need to finish up our
  // computations by computing the source signals, the source signal mixing
  // matrix, the unmixing matrix that will unmix the original, nonwhitened
  // observations, and the means of the source signals.
  //////////////////////////////////////////////////////////////////////////////

  // _d_W[3] holds the accumulated rotation matrix.
  // _d_W[1] holds the whitening matrix.
  // _d_W[0] holds the dewhitening matrix.
  // _d_Z[1] holds the whitened, zero-mean observations (transposed).

  CUBLAS_GEMM( _d_Z[0], _d_Z[1], _d_W[3] );    // _d_Z[0] <- the transposed sources
  CUBLAS_GEMM_TN( _d_W[2], _d_W[3], _d_W[1] ); // _d_W[2] <- the unmixing matrix
  CUBLAS_GEMM( _d_W[1], _d_W[0], _d_W[3] );    // _d_W[1] <- the mixing matrix

  // Copy the results from the device back to the host.
  hipMemcpy( _h_Z.elem, _d_Z[0].elem, sizeof(NUMTYPE) * _h_Z.ld * _h_Z.lag,
              hipMemcpyDeviceToHost );
  hipMemcpy( W->elem, _d_W[2].elem, sizeof(NUMTYPE) * W->rows * W->cols,
              hipMemcpyDeviceToHost );
  hipMemcpy( A->elem, _d_W[1].elem, sizeof(NUMTYPE) * A->rows * A->cols,
              hipMemcpyDeviceToHost );

  // Transpose the calculated source signals so they fit in the S matrix.
  for (col = 0; col < S->cols; col++) {
    for( row = 0; row < S->rows; row++) {
      s_i = col * S->ld + row;
      z_i = row * _h_Z.ld + col;
      S->elem[s_i] = _h_Z.elem[z_i];
    }
  }

  // Find the source signal means.
  GEMV( mu_S, *W, _mu_X );

  //////////////////////////////////////////////////////////////////////////////
  // We're done!
  //////////////////////////////////////////////////////////////////////////////

  return sweeps;
}
